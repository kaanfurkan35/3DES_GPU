#include "hip/hip_runtime.h"
﻿/*****************************************************************************
KAAN FURKAN ALTINOK 2030021 - GÖKBERK AFŞİN PEKER 2094357
3DES ON GPU IMPLEMENTATION
Course : Applied Parallel Programming on GPU
Lecturer: Dr.Alptekin Temizel

This code read inputs from ascii character (input.txt), processes 3DES on GPU and outputs the encyrpted hexadecimal formatted text to encyrpted_hex.txt 
*******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
//Custom include
#include "conversion_utils.h"

#define BYTE char
#define ITER_COUNT 1

//Uncomment in order to see the results in console, for a better console view do ITER_COUNT 1 & use a small sized plaintext
#define PRINT

//Kernel to create 3*16 keys
__global__ void genKeysCuda(const BYTE* __restrict__  dev_key1, const BYTE* __restrict__  dev_key2, const BYTE* __restrict__  dev_key3, const BYTE* __restrict__  dev_pc_1, const BYTE* __restrict__ dev_shift_keys, const BYTE* __restrict__  dev_pc_2 , BYTE* __restrict__ dev_round_key1 , BYTE* __restrict__ dev_round_key2, BYTE* __restrict__ dev_round_key3)
{
	BYTE reg_temp;
	BYTE shift_ind;
	__shared__ BYTE temp[56] ;
	const BYTE* key;

	unsigned int tid = threadIdx.x;

	//different blocks for different key sets
	if (blockIdx.x == 0)	    { key = dev_key1;}
	else if (blockIdx.x == 1)	{ key = dev_key2;}
	else if (blockIdx.x == 2)	{ key = dev_key3;}

	//initial permutation to register
	reg_temp = key[dev_pc_1[tid] - 1];

	//loop starts
#pragma unroll
	for (BYTE i = 0; i < 16; i++)
	{
		shift_ind = tid-dev_shift_keys[i];

		//index adjustment for shift. since there is left and right portions, indexes needs careful adjustment
		if (shift_ind == 27) { shift_ind = 55; }
		else if (shift_ind == -1) { shift_ind = 27; }
		else if (shift_ind == 26 && dev_shift_keys[i] == 2) { shift_ind = 54; }
		else if (shift_ind == -2 ) { shift_ind = 26; }

		//automatic shift in parallel
		temp[shift_ind] = reg_temp;

		//wait for all threads
		__syncthreads();

		//save result to register for further use
		reg_temp = temp[tid];

		//final permutation and write back the resulting keys
		if (tid < 48)
		{
			if (blockIdx.x == 0)
			{
				*(dev_round_key1 + i *48 + tid) = temp[dev_pc_2[tid] - 1];
				continue;
			}
			else if (blockIdx.x == 1)
			{
				*(dev_round_key2 + i * 48 + tid) = temp[dev_pc_2[tid] - 1];
				continue;
			}
			else if (blockIdx.x == 2)
			{
				*(dev_round_key3 + i * 48 + tid) = temp[dev_pc_2[tid] - 1];
				continue;
			}
		}

	}
}

//Kernel for encyrption and decyrption purposes CHECK SYNCTHREADS
__global__ void encyrptCuda (const BYTE* __restrict__ dev_key, const BYTE* __restrict__  dev_pt, BYTE* __restrict__ dev_ct, const BYTE* __restrict__ dev_initperm, const BYTE* __restrict__ dev_expd, const BYTE* __restrict__ dev_s, const BYTE* __restrict__ dev_per, const BYTE* __restrict__ dev_finalper)
{
	//Temporary register variable
	BYTE reg_temp;

	//Shared variables
	__shared__ BYTE temp[64];
	__shared__ BYTE temp_loop[48];

	//Thread id calcs
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tid = threadIdx.x;

	//initial permutation to shared mem
	temp[tid] = dev_pt[blockIdx.x * blockDim.x + (dev_initperm[tid] - 1)];
	__syncthreads();

	//loop starts
#pragma unroll
	for (BYTE i = 0; i < 16; i++)
	{
		if (tid < 48)
		{
			//Expansion D-box permutation and XOR with round key in one line
			temp_loop[tid] = dev_key[48 * i + tid] ^ temp[dev_expd[tid] - 1 + 32];
		}
		__syncthreads();

		//S-box operations, compute row and col, get the value from matix, create 4 bit output for each 6bit. Middle 4 bits create col, first and last bit create row.
		if (tid < 8)
		{
			int row = 2 * temp_loop[tid * 6] + temp_loop[tid * 6 + 5];
			int col = 8 * temp_loop[tid * 6 + 1] + 4 * temp_loop[tid * 6 + 2] + 2 * temp_loop[tid * 6 + 3] + temp_loop[tid * 6 + 4];
			BYTE val = dev_s[col + 16 * (row + 4 * tid)];
			temp_loop[tid * 4] = val / 8; val = val % 8;
			temp_loop[(tid * 4) + 1] = val / 4; val = val % 4;
			temp_loop[(tid * 4) + 2] = val / 2; val = val % 2;
			temp_loop[(tid * 4) + 3] = val;
		}
		__syncthreads();

		//Straight P-box
		if (tid < 32)
		{
			//Permutation and XORing it with left portion of plaintext
			reg_temp = temp[tid] ^ temp_loop[dev_per[tid] - 1];
			
			//Swap operations
			if (i == 15)
			{
				//no swap at last step
				temp[tid] = reg_temp;
				goto out;
			}
			//swap left and right, change right portion with XORed data
			temp[tid] = temp[tid + 32];
			temp[tid + 32] = reg_temp;
		}
		__syncthreads();
	}
out:
	__syncthreads();
	//Final permutation
	dev_ct[idx] = temp[dev_finalper[tid] - 1];
}

// Helper function for using CUDA to create 3DES keys.
hipError_t keyGen(BYTE* key1, BYTE* key2, BYTE* key3, BYTE round_key1[][48], BYTE round_key2[][48], BYTE round_key3[][48])
{
	//Cuda pointers
	BYTE* dev_key1 = nullptr;
	BYTE* dev_key2 = nullptr;
	BYTE* dev_key3 = nullptr;
	BYTE* dev_round_key1 = nullptr;
	BYTE* dev_round_key2 = nullptr;
	BYTE* dev_round_key3 = nullptr;
	BYTE* dev_pc_1 = nullptr;
	BYTE* dev_shift_keys = nullptr;
	BYTE* dev_pc_2 = nullptr;

    hipError_t cudaStatus;

	//Permuted choice table
	BYTE pc_1[56] =
	{ 57,49,41,33,25,17,9,
		1,58,50,42,34,26,18,
		10,2,59,51,43,35,27,
		19,11,3,60,52,44,36,
		63,55,47,39,31,23,15,
		7,62,54,46,38,30,22,
		14,6,61,53,45,37,29,
		21,13,5,28,20,12,4
	};

	//Shift table
	BYTE shift_keys[16] =
	{ 1, 1, 2, 2,
		2, 2, 2, 2,
		1, 2, 2, 2,
		2, 2, 2, 1
	};

	//Key Compression Table
	BYTE pc_2[48] =
	{ 14,17,11,24,1,5,
		3,28,15,6,21,10,
		23,19,12,4,26,8,
		16,7,27,20,13,2,
		41,52,31,37,47,55,
		30,40,51,45,33,48,
		44,49,39,56,34,53,
		46,42,50,36,29,32
	};

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three keys and matrices
    cudaStatus = hipMalloc((void**)&dev_key1, 64 * sizeof(BYTE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_key2, 64 * sizeof(BYTE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_key3, 64 * sizeof(BYTE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_pc_1, 56 * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_shift_keys, 16 * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_pc_2, 48 * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_round_key1, 16 * 48 * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_round_key2, 16 * 48 * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_round_key3, 16 * 48 * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    // Copy data from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_key1, key1, 64 * sizeof(BYTE), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_key2, key2, 64 * sizeof(BYTE), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_key3, key3, 64 * sizeof(BYTE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_pc_1, pc_1, 56 * sizeof(BYTE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_shift_keys, shift_keys, 16 * sizeof(BYTE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_pc_2, pc_2, 48 * sizeof(BYTE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemset(dev_round_key1, 0, 16 * 48 * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}

	cudaStatus = hipMemset(dev_round_key2, 0, 16 * 48 * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}

	cudaStatus = hipMemset(dev_round_key3, 0, 16 * 48 * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}

    // Launch a kernel on the GPU. A block for each key set. 56 threads each block
    genKeysCuda<<<3, 56>>>(dev_key1, dev_key2, dev_key3, dev_pc_1, dev_shift_keys, dev_pc_2, dev_round_key1, dev_round_key2, dev_round_key3);


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy resulting keys from GPU buffer to host memory.
    cudaStatus = hipMemcpy(round_key1, dev_round_key1, 16 * 48 * sizeof(BYTE), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(round_key2, dev_round_key2, 16 * 48 * sizeof(BYTE), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(round_key3, dev_round_key3, 16 * 48 * sizeof(BYTE), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

#ifdef PRINT
	//Print the keys
	printf("KEY SET 1:\n");
	for (int i = 0; i < 16; i++) {
		printf("%d.key:  ", i);
		bin48_to_hex(round_key1[i]);
		printf("\n");
	}
	printf("\nKEY SET 2:\n");
	for (int i = 0; i < 16; i++) {
		printf("%d.key:  ", i);
		bin48_to_hex(round_key2[i]);
		printf("\n");
	}
	printf("\nKEY SET 3:\n");
	for (int i = 0; i < 16; i++) {
		printf("%d.key:  ", i);
		bin48_to_hex(round_key3[i]);
		printf("\n");
	}
#endif

Error:
    hipFree(dev_key3);
    hipFree(dev_key1);
    hipFree(dev_key2);
	hipFree(dev_round_key1);
	hipFree(dev_round_key2);
	hipFree(dev_round_key3);
	hipFree(dev_pc_1);
	hipFree(dev_shift_keys);
	hipFree(dev_pc_2);

    return cudaStatus;
}

// Helper function for using CUDA to process 3DES plaintext.
hipError_t encrypt(BYTE* pt, BYTE round_key[16][48], BYTE* cipher_text, int ptblockSize)
{
	//Cuda pointers
	BYTE* dev_key = nullptr;
	BYTE* dev_pt = nullptr;
	BYTE* dev_ct = nullptr;
	BYTE* dev_initperm = nullptr;
	BYTE* dev_expd = nullptr;
	BYTE* dev_s = nullptr;
	BYTE* dev_per = nullptr;
	BYTE* dev_finalper = nullptr;
	hipError_t cudaStatus;

	//Initial Permutation
	BYTE initial_perm[64] =
	{ 58,50,42,34,26,18,10,2,
		60,52,44,36,28,20,12,4,
		62,54,46,38,30,22,14,6,
		64,56,48,40,32,24,16,8,
		57,49,41,33,25,17,9,1,
		59,51,43,35,27,19,11,3,
		61,53,45,37,29,21,13,5,
		63,55,47,39,31,23,15,7
	};

	//Expansion D-box Table
	BYTE exp_d[48] =
	{ 32,1,2,3,4,5,4,5,
		6,7,8,9,8,9,10,11,
		12,13,12,13,14,15,16,17,
		16,17,18,19,20,21,20,21,
		22,23,24,25,24,25,26,27,
		28,29,28,29,30,31,32,1
	};

	//S-box Table, total 8 s-boxes
	BYTE s[8][4][16] =
	{ {
		14,4,13,1,2,15,11,8,3,10,6,12,5,9,0,7, //0
		0,15,7,4,14,2,13,1,10,6,12,11,9,5,3,8,
		4,1,14,8,13,6,2,11,15,12,9,7,3,10,5,0,
		15,12,8,2,4,9,1,7,5,11,3,14,10,0,6,13
	},
	{
		15,1,8,14,6,11,3,4,9,7,2,13,12,0,5,10, //1
		3,13,4,7,15,2,8,14,12,0,1,10,6,9,11,5,
		0,14,7,11,10,4,13,1,5,8,12,6,9,3,2,15,
		13,8,10,1,3,15,4,2,11,6,7,12,0,5,14,9
	},
	{
		10,0,9,14,6,3,15,5,1,13,12,7,11,4,2,8, //2
		13,7,0,9,3,4,6,10,2,8,5,14,12,11,15,1,
		13,6,4,9,8,15,3,0,11,1,2,12,5,10,14,7,
		1,10,13,0,6,9,8,7,4,15,14,3,11,5,2,12
	},
	{
		7,13,14,3,0,6,9,10,1,2,8,5,11,12,4,15, //3
		13,8,11,5,6,15,0,3,4,7,2,12,1,10,14,9,
		10,6,9,0,12,11,7,13,15,1,3,14,5,2,8,4,
		3,15,0,6,10,1,13,8,9,4,5,11,12,7,2,14
	},
	{
		2,12,4,1,7,10,11,6,8,5,3,15,13,0,14,9, //4
		14,11,2,12,4,7,13,1,5,0,15,10,3,9,8,6,
		4,2,1,11,10,13,7,8,15,9,12,5,6,3,0,14,
		11,8,12,7,1,14,2,13,6,15,0,9,10,4,5,3
	},
	{
		12,1,10,15,9,2,6,8,0,13,3,4,14,7,5,11, //5
		10,15,4,2,7,12,9,5,6,1,13,14,0,11,3,8,
		9,14,15,5,2,8,12,3,7,0,4,10,1,13,11,6,
		4,3,2,12,9,5,15,10,11,14,1,7,6,0,8,13
	},
	{
		4,11,2,14,15,0,8,13,3,12,9,7,5,10,6,1, //6
		13,0,11,7,4,9,1,10,14,3,5,12,2,15,8,6,
		1,4,11,13,12,3,7,14,10,15,6,8,0,5,9,2,
		6,11,13,8,1,4,10,7,9,5,0,15,14,2,3,12
	},
	{
		13,2,8,4,6,15,11,1,10,9,3,14,5,0,12,7, //7
		1,15,13,8,10,3,7,4,12,5,6,11,0,14,9,2,
		7,11,4,1,9,12,14,2,0,6,10,13,15,3,5,8,
		2,1,14,7,4,10,8,13,15,12,9,0,3,5,6,11
	} };

	//Straight Permutation Table
	BYTE per[32] =
	{ 16,7,20,21,
		29,12,28,17,
		1,15,23,26,
		5,18,31,10,
		2,8,24,14,
		32,27,3,9,
		19,13,30,6,
		22,11,4,25
	};

	//Final Permutation Table
	BYTE final_perm[64] =
	{ 40,8,48,16,56,24,64,32,
		39,7,47,15,55,23,63,31,
		38,6,46,14,54,22,62,30,
		37,5,45,13,53,21,61,29,
		36,4,44,12,52,20,60,28,
		35,3,43,11,51,19,59,27,
		34,2,42,10,50,18,58,26,
		33,1,41,9,49,17,57,25
	};

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)&dev_key, 16 * 48 * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_pt, 64 * ptblockSize *sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_ct, 64 * ptblockSize * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_initperm, 64 * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_expd, 48 * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_s, 8 * 4 * 16 * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_per, 32 * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_finalper, 64 * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input data from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_key, round_key, 16 * 48 * sizeof(BYTE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_pt, pt, 64 * ptblockSize * sizeof(BYTE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemset(dev_ct, 0, 64 * ptblockSize * sizeof(BYTE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_initperm, initial_perm, 64 * sizeof(BYTE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_expd, exp_d, 48 * sizeof(BYTE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_s, s, 8 * 4 * 16 * sizeof(BYTE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_per, per, 32 * sizeof(BYTE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_finalper, final_perm, 64 * sizeof(BYTE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU, each 64 bit plaintext block counts as 1 block and each block has 64 threads due to 64 bit plaintext processing
	encyrptCuda << <ptblockSize, 64 >> > (dev_key, dev_pt, dev_ct, dev_initperm, dev_expd, dev_s, dev_per, dev_finalper);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Encrypt launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching encrypt kernel!\n", cudaStatus);
		goto Error;
	}

	// Copy ciphertext from GPU buffer to host memory.
	cudaStatus = hipMemcpy(cipher_text, dev_ct, 64 * ptblockSize* sizeof(BYTE), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_key);
	hipFree(dev_pt);
	hipFree(dev_ct);
	hipFree(dev_initperm);
	hipFree(dev_expd);
	hipFree(dev_s);
	hipFree(dev_per);
	hipFree(dev_finalper);

	return cudaStatus;
}

hipError_t decrypt(BYTE* pt, BYTE round_key[16][48], BYTE* cipher_text, int ptblockSize) {

	BYTE rev_key[16][48];

	//Reverse the original key
	for (int i = 0; i < 16; i++)
	{
		for (int j = 0; j < 48; j++)
		{
			rev_key[15 - i][j] = round_key[i][j];
		}
	}
	//Call encryption
	return encrypt(pt, rev_key, cipher_text, ptblockSize);
}

//Function that reads the formatted text. The structure of txt can be found in README.txt
void readText(BYTE* size_arr, float &size, BYTE* key1_int, BYTE* key2_int, BYTE* key3_int, BYTE* &pt_int)
{
	//Please refer to report for input text format which includes plaintext, keys and plaintext block size
	int data;
	FILE* file_new;
	file_new = fopen("input.txt", "r");
	if (file_new) {
		int row_ctr;
		int column_ctr = 0;

		for (row_ctr = 0; row_ctr < 5; row_ctr++) {
			//Obtain the size as integer from txt
			if (row_ctr == 4) 
			{
				float decimal_mul = 1.f;
				int temp = column_ctr;
				//Reverse the array
				for (int i = 0; i < temp - 1; i++) {
					int t = size_arr[i];
					size_arr[i] = size_arr[temp - 1];
					size_arr[temp - 1] = t;
					temp--;
				}

				//Convert to decimal int
				for (int i = 0; i < column_ctr; i++)
				{
					size = size + size_arr[i] * decimal_mul;
					decimal_mul *= 10;
				}
			}
			column_ctr = 0;
			//Read char by char until new line
			while ((data = getc(file_new)) != '\n') {
				if (row_ctr == 0) key1_int[column_ctr++] = data;
				else if (row_ctr == 1) key2_int[column_ctr++] = data;
				else if (row_ctr == 2) key3_int[column_ctr++] = data;
				else if (row_ctr == 3)
				{
					//Ascii to int
					size_arr[column_ctr++] = (int)(data - '0');
				}
				else
				{
					//Allocate only once for not converted plaintext, it will be converted to bit array later
					for (static bool first = true; first; first = false)
					{
						pt_int = (BYTE*)malloc(8 * (int)size * sizeof(BYTE));
					}
					pt_int[column_ctr++] = data;
				}
			}
		}
	}
	fclose(file_new);
}

void write_text(BYTE* input, int size, const char *str ) {
	int buf;
	FILE* fp;
	fp = fopen(str, "w");
	for (int j = 0; j < 16 * size; j++) {
		buf = 8 * input[j * 4] + 4 * input[j * 4 + 1] + 2 * input[j * 4 + 2] + input[j * 4 + 3];
		//printf("%01X", buf);
		fprintf(fp, "%X", buf);
	}
	fclose(fp);
}

int main()
{
	//Variables
	float size = 0.f;//holds plaintext block size which consists of 64bits
	BYTE* plain_text = nullptr;//plaintext
	//For encryption
	BYTE* cipher1_text = nullptr;//ciphertext after first encyrption
	BYTE* cipher2_text = nullptr;//ciphertext after second decyrption
	BYTE* cipher3_text = nullptr;//ciphertext after third encyrption
	//For decryption
	BYTE* deciphered1_text = nullptr;//ciphertext after first decyrption
	BYTE* deciphered2_text = nullptr;//ciphertext after second encyrption
	BYTE* deciphered3_text = nullptr;//ciphertext after third decyrption

	BYTE round_keys1[16][48];//round keys
	BYTE round_keys2[16][48];//round keys
	BYTE round_keys3[16][48];//round keys

	//Temporary items, used only for conversions from txt
	BYTE* pt_int = nullptr;//plaintext read from file
	BYTE key1_int[16];//key read from file
	BYTE key2_int[16];//key read from file
	BYTE key3_int[16];//key read from file
	BYTE size_arr[16];
	BYTE key1[64];//base key
	BYTE key2[64];//base key
	BYTE key3[64];//base key

	//Read the txt file to related pointers
	readText(size_arr, size, key1_int, key2_int, key3_int, pt_int);

	//Allocate buffers dynamically
	cipher1_text = (BYTE*)malloc(64 * (int)size * sizeof(BYTE));
	cipher2_text = (BYTE*)malloc(64 * (int)size * sizeof(BYTE));
	cipher3_text = (BYTE*)malloc(64 * (int)size * sizeof(BYTE));
	deciphered1_text = (BYTE*)malloc(64 * (int)size * sizeof(BYTE));
	deciphered2_text = (BYTE*)malloc(64 * (int)size * sizeof(BYTE));
	deciphered3_text = (BYTE*)malloc(64 * (int)size * sizeof(BYTE));
	plain_text = (BYTE*)malloc(64 * (int)size * sizeof(BYTE));

	//Hex to binary for input arrays
	char_to_bin(pt_int, plain_text, (int)size);
	hex_to_bin(key1_int, key1);
	hex_to_bin(key2_int, key2);
	hex_to_bin(key3_int, key3);
	hipError_t cudaStatus;

	//Call key generation
	for (int i = 0; i < ITER_COUNT; i++) {
		cudaStatus = keyGen(key1, key2, key3, round_keys1, round_keys2, round_keys3);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "key generation failed!");
			return 1;
		}
	}
	//3DES Encryption
	for (int i = 0; i < ITER_COUNT; i++) {
		cudaStatus = encrypt(plain_text, round_keys1, cipher1_text, (int)size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "encyrption1 %d failed!", i);
			return 1;
		}

		cudaStatus = decrypt(cipher1_text, round_keys2, cipher2_text, (int)size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "decyrption2 failed!");
			return 1;
		}

		cudaStatus = encrypt(cipher2_text, round_keys3, cipher3_text, (int)size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "encyrption3 failed!");
			return 1;
		}
	}

	//3DES Decryption to get the original plaintext
	for (int i = 0; i < ITER_COUNT; i++) {
		cudaStatus = decrypt(cipher3_text, round_keys3, deciphered3_text, (int)size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "decyrption4 failed!");
			return 1;
		}

		cudaStatus = encrypt(deciphered3_text, round_keys2, deciphered2_text, (int)size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "encyrption5 failed!");
			return 1;
		}

		cudaStatus = decrypt(deciphered2_text, round_keys1, deciphered1_text, (int)size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "decyrption6 failed!");
			return 1;
		}
	}
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	//Print the results
#ifdef PRINT
	printf("\nPlaint text in ASCII:\n");
	bin64_to_hex(plain_text, (int)size);
	printf("\n\nEncyrpted text step 1:\n");
	bin64_to_hex(cipher1_text, (int)size);
	printf("\n\nDecyrpted text step 2:\n");
	bin64_to_hex(cipher2_text, (int)size);
	printf("\n\nFinal encyrpted text step 3:\n");
	bin64_to_hex(cipher3_text, (int)size);
	printf("\n\nDecyrpted text step 1:\n");
	bin64_to_hex(deciphered3_text, (int)size);
	printf("\n\nEncyrpted text step 2:\n");
	bin64_to_hex(deciphered2_text, (int)size);
	printf("\n\nFinal decyrpted text step 3:\n");
	bin64_to_hex(deciphered1_text, (int)size);
	printf("\n\n");
#endif //PRINT

	//Output the plaintext hex values and encyrpted hex values to plain_hex.txt & encyrpted_hex.txt 
	write_text(plain_text, (int)size, "plain_hex.txt");
	write_text(cipher3_text, (int)size, "encyrpted_hex.txt");


	//Free to prevent memory leak
	free(pt_int);
	free(plain_text);
	free(cipher1_text);
	free(cipher2_text);
	free(cipher3_text);
	free(deciphered1_text);
	free(deciphered2_text);
	free(deciphered3_text);

	return 0;
}
